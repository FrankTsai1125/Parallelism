#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#include <lodepng.h>
#include <hip/hip_runtime.h>

#define pi 3.1415926535897932384626433832795

// Custom vector types and operations for CUDA
struct vec2 {
    float x, y;
    __host__ __device__ vec2() : x(0), y(0) {}
    __host__ __device__ vec2(float x, float y) : x(x), y(y) {}
    __host__ __device__ vec2(float v) : x(v), y(v) {}
};

struct vec3 {
    float x, y, z;
    __host__ __device__ vec3() : x(0), y(0), z(0) {}
    __host__ __device__ vec3(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ vec3(float v) : x(v), y(v), z(v) {}
};

// Vector operations
__host__ __device__ inline vec2 operator+(const vec2& a, const vec2& b) {
    return vec2(a.x + b.x, a.y + b.y);
}

__host__ __device__ inline vec2 operator*(const vec2& a, float s) {
    return vec2(a.x * s, a.y * s);
}

__host__ __device__ inline vec2 operator*(float s, const vec2& a) {
    return vec2(a.x * s, a.y * s);
}

__host__ __device__ inline vec2 operator/(const vec2& a, float s) {
    return vec2(a.x / s, a.y / s);
}

__host__ __device__ inline vec3 operator+(const vec3& a, const vec3& b) {
    return vec3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ inline vec3 operator-(const vec3& a, const vec3& b) {
    return vec3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ inline vec3 operator*(const vec3& a, float s) {
    return vec3(a.x * s, a.y * s, a.z * s);
}

__host__ __device__ inline vec3 operator*(float s, const vec3& a) {
    return vec3(a.x * s, a.y * s, a.z * s);
}

__host__ __device__ inline vec3 operator*(const vec3& a, const vec3& b) {
    return vec3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__host__ __device__ inline vec3 operator/(const vec3& a, float s) {
    return vec3(a.x / s, a.y / s, a.z / s);
}

__host__ __device__ inline void operator+=(vec3& a, const vec3& b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

// Vector functions
__host__ __device__ inline float dot(const vec3& a, const vec3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ inline vec3 cross(const vec3& a, const vec3& b) {
    return vec3(a.y * b.z - a.z * b.y,
                a.z * b.x - a.x * b.z,
                a.x * b.y - a.y * b.x);
}

__host__ __device__ inline float length(const vec3& v) {
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__host__ __device__ inline vec3 normalize(const vec3& v) {
    float len = length(v);
    if (len > 0.0f) {
        return vec3(v.x / len, v.y / len, v.z / len);
    }
    return vec3(0.0f, 0.0f, 0.0f);
}

__host__ __device__ inline float vmin(float a, float b) {
    return a < b ? a : b;
}

__host__ __device__ inline float vmax(float a, float b) {
    return a > b ? a : b;
}

__host__ __device__ inline float clamp(float x, float minVal, float maxVal) {
    return vmin(vmax(x, minVal), maxVal);
}

__host__ __device__ inline vec3 vmin(const vec3& a, float b) {
    return vec3(vmin(a.x, b), vmin(a.y, b), vmin(a.z, b));
}

__host__ __device__ inline vec3 clamp(const vec3& v, float minVal, float maxVal) {
    return vec3(clamp(v.x, minVal, maxVal),
                clamp(v.y, minVal, maxVal),
                clamp(v.z, minVal, maxVal));
}

__host__ __device__ inline vec3 vcos(const vec3& v) {
    return vec3(cosf(v.x), cosf(v.y), cosf(v.z));
}

__host__ __device__ inline vec3 vpow(const vec3& v, const vec3& p) {
    return vec3(powf(v.x, p.x), powf(v.y, p.y), powf(v.z, p.z));
}

// GLM-like swizzle functions
__host__ __device__ inline vec3 xyy(const vec2& v) { return vec3(v.x, v.y, v.y); }
__host__ __device__ inline vec3 yxy(const vec2& v) { return vec3(v.y, v.x, v.y); }
__host__ __device__ inline vec3 yyx(const vec2& v) { return vec3(v.y, v.y, v.x); }

// Constants
constexpr int AA = 3;
constexpr int MD_ITER = 24;
constexpr int RAY_STEP = 10000;
constexpr int SHADOW_STEP = 1500;
__constant__ float power = 8.0f;
__constant__ float step_limiter = 0.2f;
__constant__ float ray_multiplier = 0.1f;
__constant__ float bailout = 2.0f;
__constant__ float eps = 0.0005f;
__constant__ float FOV = 1.5f;
__constant__ float far_plane = 100.0f;

// Store camera and resolution in constant memory
__constant__ float d_camera_pos_x, d_camera_pos_y, d_camera_pos_z;
__constant__ float d_target_pos_x, d_target_pos_y, d_target_pos_z;
__constant__ float d_iResolution_x, d_iResolution_y;
__constant__ float3 d_cf;
__constant__ float3 d_cs;
__constant__ float3 d_cu;
__constant__ float3 d_sd;

// Mandelbulb distance function
__device__ float md(const vec3& p, float& trap) {
    vec3 v = p;
    float dr = 1.0;
    float r = length(v);
    trap = r;

    for (int i = 0; i < MD_ITER; ++i) {
        float inv_r = fmaxf(r, 1e-6f);
        inv_r = 1.0f / inv_r;

        float theta = atan2f(v.y, v.x) * power;
        float phi = asinf(fmaxf(fminf(v.z * inv_r, 1.0f), -1.0f)) * power;

        float sinTheta, cosTheta;
        float sinPhi, cosPhi;
        sincosf(theta, &sinTheta, &cosTheta);
        sincosf(phi, &sinPhi, &cosPhi);

        float r2 = r * r;
        float r4 = r2 * r2;
        float r6 = r4 * r2;
        float r7 = r6 * r;
        float r8 = r4 * r4;

        dr = power * r7 * dr + 1.0f;
        v = p + r8 * vec3(cosTheta * cosPhi,
                          cosPhi * sinTheta,
                          -sinPhi);

        trap = vmin(trap, r);

        r = length(v);
        if (r > bailout) break;
    }
    return 0.5 * logf(r) * r / dr;
}

// Scene mapping - 90 degree rotation around X-axis: (x,y,z) -> (x,-z,y)
__device__ float map(const vec3& p, float& trap, int& ID) {
    vec3 rp = vec3(p.x, -p.z, p.y);  // 90 deg rotation, no trig functions needed!
    ID = 1;
    return md(rp, trap);
}

__device__ float map(const vec3& p) {
    float dmy;
    int dmy2;
    return map(p, dmy, dmy2);
}

// Palette function
__device__ vec3 pal(float t, vec3 a, vec3 b, vec3 c, vec3 d) {
    return a + b * vcos(2.0 * pi * (c * t + d));
}

// Soft shadow
__device__ float softshadow(const vec3& ro, const vec3& rd, float k) {
    float res = 1.0f;
    float t = 0.0f;
    for (int i = 0; i < SHADOW_STEP; ++i) {
        float h = map(ro + rd * t);
        if (t > 0.0f) {
            float candidate = k * h / t;
            res = fminf(res, candidate);
            if (res <= 0.02f) return 0.02f;
            if (res >= 0.99f && h >= 0.5f && t >= 5.0f) return 1.0f;
        }
        t += clamp(h, 0.001f, step_limiter);
        if (t > far_plane) break;
    }
    return clamp(res, 0.02f, 1.0f);
}

// Calculate surface normal
__device__ vec3 calcNor(const vec3& p) {
    vec2 e = vec2(eps, 0.0f);
    return normalize(vec3(
        map(p + xyy(e)) - map(p - xyy(e)),
        map(p + yxy(e)) - map(p - yxy(e)),
        map(p + yyx(e)) - map(p - yyx(e))
    ));
}

// Ray tracing
__device__ float trace(const vec3& ro, const vec3& rd, float& trap, int& ID) {
    float t = 0;
    float len = 0;

    for (int i = 0; i < RAY_STEP; ++i) {
        len = map(ro + rd * t, trap, ID);
        if (fabsf(len) < eps || t > far_plane) break;
        t += len * ray_multiplier;
    }
    return t < far_plane ? t : -1.0;
}

// Main kernel
__launch_bounds__(256, 4)
__global__ void render_kernel(unsigned char* image, unsigned int width, unsigned int height) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= height || j >= width) return;

    // Pre-compute values that don't change across AA samples
    const vec3 camera_pos = vec3(d_camera_pos_x, d_camera_pos_y, d_camera_pos_z);
    const vec3 cf = vec3(d_cf.x, d_cf.y, d_cf.z);
    const vec3 cs = vec3(d_cs.x, d_cs.y, d_cs.z);
    const vec3 cu = vec3(d_cu.x, d_cu.y, d_cu.z);
    const vec3 sd = vec3(d_sd.x, d_sd.y, d_sd.z);
    const vec3 sc = vec3(1., .9, .717);
    const vec3 ambc = vec3(0.3);
    const float gloss = 32.0;

    vec3 fcol(0.0);

    const float invAA = 1.0f / (float)AA;
    const vec2 resolution = vec2(d_iResolution_x, d_iResolution_y);
    const vec2 screen_origin = vec2(-d_iResolution_x, -d_iResolution_y);
    const vec2 pixel_base = vec2((float)j, (float)i);

    for (int m = 0; m < AA; ++m) {
        for (int n = 0; n < AA; ++n) {
            vec2 p = vec2((float)m, (float)n) * invAA;
            vec2 uv = (screen_origin + 2.0f * (pixel_base + p)) / resolution.y;
            uv.y *= -1;

            vec3 rd = normalize(uv.x * cs + uv.y * cu + FOV * cf);

            float trap;
            int objID;
            float d = trace(camera_pos, rd, trap, objID);

            vec3 col(0.0);
            if (d >= 0.) {
                vec3 pos = camera_pos + rd * d;
                vec3 nr = calcNor(pos);
                vec3 hal = normalize(sd - rd);

                col = pal(trap - 0.4, vec3(0.5), vec3(0.5), vec3(1.0), vec3(0.0, 0.1, 0.2));
                
                float amb = (0.7 + 0.3 * nr.y) * (0.2 + 0.8 * clamp(0.05 * logf(trap), 0.0, 1.0));
                float sdw = softshadow(pos + 0.001 * nr, sd, 16.0);
                float dif = clamp(dot(sd, nr), 0.0, 1.0) * sdw;
                float spe = powf(clamp(dot(nr, hal), 0.0, 1.0), gloss) * dif;

                vec3 lin(0.0);
                lin += ambc * (0.05 + 0.95 * amb);
                lin += sc * dif * 0.8;
                col = col * lin;

                col = vpow(col, vec3(0.7, 0.9, 1.0));
                col += vec3(spe * 0.8);
            }

            fcol += clamp(vpow(col, vec3(0.4545)), 0.0, 1.0);
        }
    }

    fcol = fcol * (255.0 / (float)(AA * AA));

    int idx = i * width * 4 + j * 4;
    image[idx + 0] = (unsigned char)fcol.x;
    image[idx + 1] = (unsigned char)fcol.y;
    image[idx + 2] = (unsigned char)fcol.z;
    image[idx + 3] = 255;
}

void write_png(const char* filename, unsigned char* image, unsigned int width, unsigned int height) {
    unsigned error = lodepng_encode32_file(filename, image, width, height);
    if (error) printf("png error %u: %s\n", error, lodepng_error_text(error));
}

int main(int argc, char** argv) {
    assert(argc == 10);

    float camera_pos_x = atof(argv[1]);
    float camera_pos_y = atof(argv[2]);
    float camera_pos_z = atof(argv[3]);
    float target_pos_x = atof(argv[4]);
    float target_pos_y = atof(argv[5]);
    float target_pos_z = atof(argv[6]);
    unsigned int width = atoi(argv[7]);
    unsigned int height = atoi(argv[8]);
    float iResolution_x = width;
    float iResolution_y = height;

    hipMemcpyToSymbol(HIP_SYMBOL(d_camera_pos_x), &camera_pos_x, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_camera_pos_y), &camera_pos_y, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_camera_pos_z), &camera_pos_z, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_pos_x), &target_pos_x, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_pos_y), &target_pos_y, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_pos_z), &target_pos_z, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_iResolution_x), &iResolution_x, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_iResolution_y), &iResolution_y, sizeof(float));

    vec3 h_camera_pos(camera_pos_x, camera_pos_y, camera_pos_z);
    vec3 h_target_pos(target_pos_x, target_pos_y, target_pos_z);
    vec3 h_cf = normalize(h_target_pos - h_camera_pos);
    if (length(h_cf) < 1e-6f) {
        h_cf = vec3(0.0f, 0.0f, -1.0f);
    }

    vec3 up_vector(0.0f, 1.0f, 0.0f);
    vec3 side_candidate = cross(h_cf, up_vector);
    if (length(side_candidate) < 1e-4f) {
        up_vector = vec3(0.0f, 0.0f, 1.0f);
        side_candidate = cross(h_cf, up_vector);
    }

    vec3 h_cs = normalize(side_candidate);
    vec3 h_cu = normalize(cross(h_cs, h_cf));
    if (length(h_cu) < 1e-6f) {
        h_cu = vec3(0.0f, 1.0f, 0.0f);
    }

    float cam_length = length(h_camera_pos);
    vec3 h_sd = cam_length > 1e-6f ? h_camera_pos / cam_length : vec3(0.0f, 1.0f, 0.0f);

    float3 cf_const = make_float3(h_cf.x, h_cf.y, h_cf.z);
    float3 cs_const = make_float3(h_cs.x, h_cs.y, h_cs.z);
    float3 cu_const = make_float3(h_cu.x, h_cu.y, h_cu.z);
    float3 sd_const = make_float3(h_sd.x, h_sd.y, h_sd.z);

    hipMemcpyToSymbol(HIP_SYMBOL(d_cf), &cf_const, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cs), &cs_const, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_cu), &cu_const, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_sd), &sd_const, sizeof(float3));

    unsigned char* d_image;
    size_t image_size = width * height * 4 * sizeof(unsigned char);
    hipMalloc(&d_image, image_size);

    // Use 16x16 blocks to reduce per-block register pressure and improve SM occupancy
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    render_kernel<<<gridDim, blockDim>>>(d_image, width, height);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();

    unsigned char* h_image = new unsigned char[image_size];
    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);

    write_png(argv[9], h_image, width, height);

    hipFree(d_image);
    delete[] h_image;

    return 0;
}
